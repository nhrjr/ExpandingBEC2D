#ifndef CUDAREALGRID_CU__
#define CUDAREALGRID_CU__

#include <iostream>
#include <omp.h>

#include "cudarealgrid.h"
#include "realgrid.h"
#include "wrapped_cuda_functions.h"

CudaRealGrid::CudaRealGrid()
{
	dim[0] = dim[1] = dim[2] = internal = fft_dim[0] = fft_dim[1] = fft_dim[2] = 0;
	store = NULL;
}

CudaRealGrid::CudaRealGrid(uint32_t internal, uint32_t w, uint32_t h, uint32_t d)
{
	initialize(internal, w, h, d);
}

CudaRealGrid::CudaRealGrid(const CudaRealGrid &grid)
{
	initialize(internal, grid.dim[0], grid.dim[1], grid.dim[2]);
	hipMemcpy(store, grid.store, internal*fft_dim[0]*fft_dim[1]*fft_dim[2]/2*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToDevice);
}

CudaRealGrid::~CudaRealGrid()
{
	deallocate();
}

void CudaRealGrid::createPlan()
{
	hipfftResult res_forward;
	hipfftResult res_backward;

	uint32_t rank = dim[2]>1 ? 3 : ((dim[2]==1 && dim[1]>1) ? 2 : ((dim[2] == 1 && dim[1]==1 && dim[0] > 1) ? 1 : 3));
			
	int n[rank];
			
	if (rank==1)
		n[0] = dim[0];
	else if (rank ==2)
	{
		n[0] = dim[0];
		n[1] = dim[1];
	}
	else if (rank == 3)
	{
		n[0] = dim[0];
		n[1] = dim[1];
		n[2] = dim[2];
	}

	res_forward = hipfftPlanMany(&plan_forward, rank, n, NULL, 1, dim[0]*dim[1]*dim[2], NULL, 1, fft_dim[0]*fft_dim[1]*fft_dim[2]/2, HIPFFT_D2Z, internal);
	res_backward = hipfftPlanMany(&plan_backward, rank, n, NULL, 1, fft_dim[0]*fft_dim[1]*fft_dim[2]/2, NULL, 1, dim[0]*dim[1]*dim[2], HIPFFT_Z2D, internal);

	if(res_forward != HIPFFT_SUCCESS)
	{
		cout << "Creating CUFFT-Plan D2Z failed in thread " << omp_get_thread_num() << "!" << endl << "Reason:  ";
		switch(res_forward) {
			case HIPFFT_SETUP_FAILED:
				cout << "Setup failed!" << endl;
			case HIPFFT_INVALID_SIZE:
				cout << "Invalid size!" << endl;
			case HIPFFT_INVALID_TYPE:
				cout << "Invalid type!" << endl;
			case HIPFFT_ALLOC_FAILED:
				cout << "Allocation failed!" << endl;
			default:
				cout << "Unknown error!" << endl;
				break;
		}
	}
	
	if(res_backward != HIPFFT_SUCCESS)
	{
		cout << "Creating CUFFT-Plan Z2D failed in thread " << omp_get_thread_num() << "!" << endl << "Reason:  ";
		switch(res_backward) {
			case HIPFFT_SETUP_FAILED:
				cout << "Setup failed!" << endl;
			case HIPFFT_INVALID_SIZE:
				cout << "Invalid size!" << endl;
			case HIPFFT_INVALID_TYPE:
				cout << "Invalid type!" << endl;
			case HIPFFT_ALLOC_FAILED:
				cout << "Allocation failed!" << endl;
			default:
				cout << "Unknown error!" << endl;
				break;
		}
	}
	
	if (cufftSetCompatibilityMode(plan_forward, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS)
	{
		cout << "error occured while setting FFT compatibility to NATIVE for forward plan" << endl;
	}
	
	if (cufftSetCompatibilityMode(plan_backward, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS)
	{
		cout << "error occured while setting FFT compatibility to NATIVE for backward plan" << endl;
	}
}

void CudaRealGrid::initialize(uint32_t int_dim, uint32_t w, uint32_t h, uint32_t d)
{
	dim[0] = w;
	dim[1] = h;
	dim[2] = d;
	internal = int_dim;
	if (d > 1)
	{
		fft_dim[0] = w;
		fft_dim[1] = h;
		fft_dim[2] = 2*(int(d/2) + 1);
 	}
	else if (d == 1 && h > 1)
	{
		fft_dim[0] = w;
		fft_dim[1] = 2*(int(h/2) + 1);
		fft_dim[2] = 1;
	}
	else if (d == 1 && h == 1 && w > 1)
	{
		fft_dim[0] = 2*(int(w/2) + 1);
		fft_dim[1] = 1;
		fft_dim[2] = 1;
	}
	else
	{
		fft_dim[0] = 1;
		fft_dim[1] = 1;
		fft_dim[2] = 1;
	}
	
	if(hipMalloc(&store, sizeof(hipfftDoubleComplex)*fft_dim[0]*fft_dim[1]*fft_dim[2]*internal/2) != hipSuccess)
	{
		cout << "Error allocating Cuda-Memory in thread " << omp_get_thread_num() << "!" << endl;
	}
	createPlan();
}

void CudaRealGrid::deallocate()
{
	hipfftDestroy(plan_forward);
	hipfftDestroy(plan_backward);
	hipFree(store);
}

inline const char *cufft_error_string(int error_id)
{
	switch(error_id) {
		case HIPFFT_SUCCESS:					// Should never occur
			return "Success!";
		case HIPFFT_SETUP_FAILED:
			return "Setup failed!";
		case HIPFFT_INVALID_PLAN:
			return "Invalid plan!";
		case HIPFFT_INVALID_VALUE:
			return "Invalid value!";
		case HIPFFT_EXEC_FAILED:
			return "Execution on GPU failed!";
		default:
			return "Unknown error!";
	}
}

bool CudaRealGrid::fft(CudaRealGrid &i, CudaRealGrid &o, int direction)
{
	int res_forward[4] = {HIPFFT_SUCCESS, HIPFFT_SUCCESS, HIPFFT_SUCCESS, HIPFFT_SUCCESS};
	int res_backward[4] = {HIPFFT_SUCCESS, HIPFFT_SUCCESS, HIPFFT_SUCCESS, HIPFFT_SUCCESS};
	
	if (direction == HIPFFT_FORWARD)
	{
		int j = -1;
		do
		{
			j++;
			res_forward[j] = hipfftExecD2Z(i.plan_forward, (double*)i.store, o.store);
		} while((res_forward[j] != HIPFFT_SUCCESS) && (j < 3));
	}
	
	else if (direction == HIPFFT_BACKWARD)
	{
		int j = -1;
		do
		{
			j++;
			res_backward[j] = hipfftExecZ2D(i.plan_backward, i.store, (double*)o.store);
		} while((res_backward[j] != HIPFFT_SUCCESS) && (j < 3));
	}
	
	else
	{
		cout << "no valid direction chosen" << endl;
		return false;
	}
	
	if(res_forward[0] != HIPFFT_SUCCESS)
	{
		cout << "Thread " << omp_get_thread_num() << ", Direction forward" << ":" << endl;
		for(int k = 0; k < 4; k++)
		{
			if(res_forward[k] != HIPFFT_SUCCESS)
				cout << "Warning: Try " << k << " of fourier transform failed: " << cufft_error_string(res_forward[k]) << endl;
		}
		if((res_forward[1] != HIPFFT_SUCCESS) && (res_forward[2] != HIPFFT_SUCCESS) && (res_forward[3] != HIPFFT_SUCCESS))
		{
			cout << "Error: All trys of fourier transforming failed in thread " << omp_get_thread_num() << "!" << endl;
			return false;
		}
	}
	
	if(res_backward[0] != HIPFFT_SUCCESS)
	{
		cout << "Thread " << omp_get_thread_num() << ", Direction backward" << ":" << endl;
		for(int k = 0; k < 4; k++)
		{
			if(res_backward[k] != HIPFFT_SUCCESS)
				cout << "Warning: Try " << k << " of fourier transform failed: " << cufft_error_string(res_backward[k]) << endl;
		}
		if((res_backward[1] != HIPFFT_SUCCESS) && (res_backward[2] != HIPFFT_SUCCESS) && (res_backward[3] != HIPFFT_SUCCESS))
		{
			cout << "Error: All trys of fourier transforming failed in thread " << omp_get_thread_num() << "!" << endl;
			return false;
		}
	}
	
	return true;
}

void CudaRealGrid::resize (uint32_t internal, uint32_t w, uint32_t h, uint32_t d)
{
	deallocate();
	initialize(internal, w, h, d);
}

CudaRealGrid &CudaRealGrid::operator= (const CudaRealGrid &grid)
{
	if(&grid != this)
	{
		if((internal != grid.internal) || (dim[0] != grid.dim[0]) || (dim[1] != grid.dim[1]) || (dim[2] != grid.dim[2]))
			resize(grid.internal, grid.dim[0], grid.dim[1], grid.dim[2]);

		hipMemcpy(store, grid.store, fft_dim[0]*fft_dim[1]*fft_dim[2]/2*sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);
	}
	return *this;
}

CudaRealGrid &copyHostToDevice_as_complex (CudaRealGrid &cgrid, const RealGrid &grid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing CudaRealGrid for copying!" << endl;
		cgrid.resize(grid.int_dim(), grid.width(), grid.height(), grid.depth());
	}
	double *copygrid = new double[grid.int_dim()*grid.width()*grid.height()*grid.depth()];


	memcpy_host_to_device(cgrid.store, grid.store, sizeof(complex<double>)*grid.int_dim()*grid.fft_width()*grid.fft_height()*grid.fft_depth()/2);

	
	return cgrid;
}


CudaRealGrid &copyHostToDevice3D (CudaRealGrid &cgrid, const RealGrid &grid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing CudaRealGrid for copying!" << endl;
		cgrid.resize(grid.int_dim(), grid.width(), grid.height(), grid.depth());
	}
	double *copygrid = new double[grid.int_dim()*grid.width()*grid.height()*grid.depth()];
	//remove padding in grid by copying row by row to copygrid
	for(int mu = 0; mu < grid.int_dim(); mu++)
	{
		for(int x = 0; x < grid.width(); x++)
		{
			for(int y = 0; y < grid.height(); y++)
			{
				memcpy(copygrid + mu*cgrid.width()*cgrid.height()*cgrid.depth() + x*cgrid.height()*cgrid.depth() + y*cgrid.depth(), grid.store + mu*grid.fft_width()*grid.fft_height()*grid.fft_depth() + x*grid.fft_height()*grid.fft_depth() + y*grid.fft_depth(), grid.depth()*sizeof(double));
			}
		}
	}
	//finally copy unpadded grid to CUDA device
	memcpy_host_to_device(cgrid.store, copygrid, sizeof(double)*grid.int_dim()*grid.width()*grid.height()*grid.depth());
	//clean up
	delete copygrid;
	
	return cgrid;
}

CudaRealGrid &copyHostToDevice2D (CudaRealGrid &cgrid, const RealGrid &grid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing CudaRealGrid for copying!" << endl;
		cgrid.resize(grid.int_dim(), grid.width(), grid.height(), grid.depth());
	}
	double *copygrid = new double[grid.int_dim()*grid.width()*grid.height()*grid.depth()];
	//remove padding in grid by copying row by row to copygrid
	for(int mu = 0; mu < grid.int_dim(); mu++)
	{
		for(int x = 0; x < grid.width(); x++)
		{
			memcpy(copygrid + mu*cgrid.width()*cgrid.height()*cgrid.depth() + x*cgrid.height()*cgrid.depth(), grid.store + mu*grid.fft_width()*grid.fft_height()*grid.fft_depth() + x*grid.fft_height()*grid.fft_depth(), grid.height()*sizeof(double));			
		}
	}
	//finally copy unpadded grid to CUDA device
	memcpy_host_to_device(cgrid.store, copygrid, sizeof(double)*grid.int_dim()*grid.width()*grid.height()*grid.depth());
	//clean up
	delete copygrid;
	
	return cgrid;
}

CudaRealGrid &copyHostToDevice1D (CudaRealGrid &cgrid, const RealGrid &grid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing CudaRealGrid for copying!" << endl;
		cgrid.resize(grid.int_dim(), grid.width(), grid.height(), grid.depth());
	}
	double *copygrid = new double[grid.int_dim()*grid.width()*grid.height()*grid.depth()];
	//remove padding in grid by copying row by row to copygrid
	for(int mu = 0; mu < grid.int_dim(); mu++)
	{
		memcpy(copygrid + mu*cgrid.width()*cgrid.height()*cgrid.depth(), grid.store + mu*grid.fft_width()*grid.fft_height()*grid.fft_depth(), grid.width()*sizeof(double));
	}
	//finally copy unpadded grid to CUDA device
	memcpy_host_to_device(cgrid.store, copygrid, sizeof(double)*grid.int_dim()*grid.width()*grid.height()*grid.depth());
	//clean up
	delete copygrid;
	
	return cgrid;
}

RealGrid &copyDeviceToHost_as_complex (RealGrid &grid, const CudaRealGrid &cgrid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing RealGrid for copying!" << endl;
		grid.resize(cgrid.int_dim(), cgrid.width(), cgrid.height(), cgrid.depth());
	}
	
	memcpy_device_to_host(grid.store, cgrid.store, sizeof(complex<double>)*grid.int_dim()*grid.fft_width()*grid.fft_height()*grid.fft_depth()/2);
	
	return grid;
}



RealGrid &copyDeviceToHost3D (RealGrid &grid, const CudaRealGrid &cgrid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing RealGrid for copying!" << endl;
		grid.resize(cgrid.int_dim(), cgrid.width(), cgrid.height(), cgrid.depth());
	}
	
	double *copygrid = new double[cgrid.int_dim()*cgrid.width()*cgrid.height()*cgrid.depth()];
	//copy unpadded grid from CUDA device to copygrid on host
	memcpy_device_to_host(copygrid, cgrid.store, cgrid.int_dim()*cgrid.width()*cgrid.height()*cgrid.depth()*sizeof(double));
	//add padding by copying row by row to grid.store
	for(int mu = 0; mu < grid.int_dim(); mu++)
	{
		for(int x = 0; x < grid.width(); x++)
		{
			for(int y = 0; y < grid.height(); y++)
			{
				memcpy(grid.store + mu*grid.fft_width()*grid.fft_height()*grid.fft_depth() + x*grid.fft_height()*grid.fft_depth() + y*grid.fft_depth(), copygrid + mu*cgrid.width()*cgrid.height()*cgrid.depth() + x*cgrid.height()*cgrid.depth() + y*cgrid.depth(), cgrid.depth()*sizeof(double));
			}
		}
	}	
	//clean up
	delete copygrid;
	
	return grid;
}

RealGrid &copyDeviceToHost2D (RealGrid &grid, const CudaRealGrid &cgrid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing RealGrid for copying!" << endl;
		grid.resize(cgrid.int_dim(), cgrid.width(), cgrid.height(), cgrid.depth());
	}
	
	double *copygrid = new double[cgrid.int_dim()*cgrid.width()*cgrid.height()*cgrid.depth()];
	//copy unpadded grid from CUDA device to copygrid on host
	memcpy_device_to_host(copygrid, cgrid.store, cgrid.int_dim()*cgrid.width()*cgrid.height()*cgrid.depth()*sizeof(double));
	//add padding by copying row by row to grid.store
	for(int mu = 0; mu < grid.int_dim(); mu++)
	{
		for(int x = 0; x < grid.width(); x++)
		{
			memcpy(grid.store + mu*grid.fft_width()*grid.fft_height()*grid.fft_depth() + x*grid.fft_height()*grid.fft_depth(), copygrid + mu*cgrid.width()*cgrid.height()*cgrid.depth() + x*cgrid.height()*cgrid.depth(), cgrid.height()*sizeof(double));
		}
	}	
	//clean up
	delete copygrid;
	
	return grid;
}

RealGrid &copyDeviceToHost1D (RealGrid &grid, const CudaRealGrid &cgrid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing RealGrid for copying!" << endl;
		grid.resize(cgrid.int_dim(), cgrid.width(), cgrid.height(), cgrid.depth());
	}
	
	double *copygrid = new double[cgrid.int_dim()*cgrid.width()*cgrid.height()*cgrid.depth()];
	//copy unpadded grid from CUDA device to copygrid on host
	memcpy_device_to_host(copygrid, cgrid.store, cgrid.int_dim()*cgrid.width()*cgrid.height()*cgrid.depth()*sizeof(double));
	//add padding by copying row by row to grid.store
	for(int mu = 0; mu < grid.int_dim(); mu++)
	{
		memcpy(grid.store + mu*grid.fft_width()*grid.fft_height()*grid.fft_depth(), copygrid + mu*cgrid.width()*cgrid.height()*cgrid.depth(), cgrid.width()*sizeof(double));
	}	
	//clean up
	delete copygrid;
	
	return grid;
}

#endif
