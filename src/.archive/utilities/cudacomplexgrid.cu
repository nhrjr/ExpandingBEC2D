#include "hip/hip_runtime.h"
#ifndef CUDACOMPLEXGRID_CU__
#define CUDACOMPLEXGRID_CU__

#include <iostream>
#include <omp.h>

#include "cudacomplexgrid.h"
#include "complexgrid.h"
#include "wrapped_cuda_functions.h"

CudaComplexGrid::CudaComplexGrid()
{
	dim[0] = dim[1] = dim[2] = internal = 0;
	store = NULL;
}

CudaComplexGrid::CudaComplexGrid(uint32_t int_dim, uint32_t w, uint32_t h, uint32_t d)
{
	initialize(int_dim, w, h, d);
}

CudaComplexGrid::CudaComplexGrid(const CudaComplexGrid &grid)
{
	initialize(grid.internal, grid.dim[0], grid.dim[1], grid.dim[2]);
	hipMemcpy2D(store, pitch, grid.store, grid.pitch, dim[0]*dim[1]*dim[2]*sizeof(hipDoubleComplex), internal, hipMemcpyDeviceToDevice);
}

CudaComplexGrid::~CudaComplexGrid()
{
	deallocate();
}

void CudaComplexGrid::createPlan(const size_t ipitch, const size_t opitch)
{
	hipfftResult res;
			
	int rank = dim[2] > 1 ? 3 : ((dim[2]==1 && dim[1] > 1) ? 2 : ((dim[2]==1 && dim[1]==1 && dim[0]>1) ? 1 : 0));
	int n[rank];

	if (rank == 1)
	{
		n[0] = dim[0];
	}
	else if (rank == 2)
	{
		n[0] = dim[0];
		n[1] = dim[1];
	}
	else if (rank == 3)
	{
		n[0] = dim[0];
		n[1] = dim[1];
		n[2] = dim[2];
	}

	res = hipfftPlanMany(&plan, rank, n, NULL, 1, (size_t) ipitch/sizeof(hipDoubleComplex), NULL, 1, (size_t) opitch/sizeof(hipDoubleComplex), HIPFFT_Z2Z, internal);
	

	if(res != HIPFFT_SUCCESS)
	{
		cout << "Creating CUFFT-Plan failed in thread " << omp_get_thread_num() << "!" << endl << "Reason:  ";
		switch(res) {
			case HIPFFT_SETUP_FAILED:
				cout << "Setup failed!" << endl;
			case HIPFFT_INVALID_SIZE:
				cout << "Invalid size!" << endl;
			case HIPFFT_INVALID_TYPE:
				cout << "Invalid type!" << endl;
			case HIPFFT_ALLOC_FAILED:
				cout << "Allocation failed!" << endl;
			default:
				cout << "Unknown error!" << endl;
				break;
		}
	}
}

void CudaComplexGrid::initialize(uint32_t int_dim, uint32_t w, uint32_t h, uint32_t d)
{
	dim[0] = w;
	dim[1] = h;
	dim[2] = d;
	internal = int_dim;

	if(hipMallocPitch(&store, &pitch,  sizeof(hipDoubleComplex)*dim[0]*dim[1]*dim[2], internal) != hipSuccess)
	{
		cout << "Error allocating Cuda-Memory in thread " << omp_get_thread_num() << "!" << endl;
	}
	if (pitch/(sizeof(hipDoubleComplex)) != floor(pitch/(sizeof(hipDoubleComplex))))
	{
		cout << "development error: pitch for cudacomplexgrid is not an integer multiple of hipDoubleComplex. CUFFT will most likely fail" << endl;
	}
	createPlan(pitch, pitch);
}

void CudaComplexGrid::deallocate()
{
	hipfftDestroy(plan);
	hipFree(store);
}

inline const char *cufft_error_string(int error_id)
{
	switch(error_id) {
		case HIPFFT_SUCCESS:					// Should never occur
			return "Success!";
		case HIPFFT_SETUP_FAILED:
			return "Setup failed!";
		case HIPFFT_INVALID_PLAN:
			return "Invalid plan!";
		case HIPFFT_INVALID_VALUE:
			return "Invalid value!";
		case HIPFFT_EXEC_FAILED:
			return "Execution on GPU failed!";
		default:
			return "Unknown error!";
	}
}

bool CudaComplexGrid::fft(CudaComplexGrid &i, CudaComplexGrid &o, int direction)
{
	int res[4] = {HIPFFT_SUCCESS, HIPFFT_SUCCESS, HIPFFT_SUCCESS, HIPFFT_SUCCESS};
	int j = -1;

	if(i.pitch != o.pitch)
	{
		cout << "development error: input pitch does not match output pitch. out of place CUFFT will most likely fail" << endl;
	}

	do
	{
		j++;
		res[j] = hipfftExecZ2Z(i.plan, i.store, o.store, direction);
	} while((res[j] != HIPFFT_SUCCESS) && (j < 3));
	if(res[0] != HIPFFT_SUCCESS)
	{
		string dir = (direction == HIPFFT_BACKWARD) ? "inverse" : ((direction == HIPFFT_FORWARD) ? "forward" : "Invalid direction");
		cout << "Thread " << omp_get_thread_num() << ", Direction " << dir << ":" << endl;
		for(int k = 0; k < 4; k++)
		{
			if(res[k] != HIPFFT_SUCCESS)
				cout << "Warning: Try " << k << " of fourier transform failed: " << cufft_error_string(res[k]) << endl;
		}
		if((res[1] != HIPFFT_SUCCESS) && (res[2] != HIPFFT_SUCCESS) && (res[3] != HIPFFT_SUCCESS))
		{
			cout << "Error: All trys of fourier transforming failed in thread " << omp_get_thread_num() << "!" << endl;
			return false;
		}
	}
	return true;
}

void CudaComplexGrid::resize (uint32_t int_dim, uint32_t w, uint32_t h, uint32_t d)
{
	deallocate();
	initialize(int_dim, w, h, d);
}

CudaComplexGrid &CudaComplexGrid::operator= (const CudaComplexGrid &grid)
{
	if(&grid != this)
	{
		if((internal != grid.internal) || (dim[0] != grid.dim[0]) || (dim[1] != grid.dim[1]) || (dim[2] != grid.dim[2]))
			resize(grid.internal, grid.dim[0], grid.dim[1], grid.dim[2]);
		hipMemcpy2D(store, pitch, grid.store, grid.get_pitch(), dim[0]*dim[1]*dim[2]*sizeof(hipDoubleComplex), internal, hipMemcpyDeviceToDevice);
	}
	return *this;
}

CudaComplexGrid &copyHostToDevice (CudaComplexGrid &cgrid, const ComplexGrid &grid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing CudaComplexGrid for copying!" << endl;
		cgrid.resize(grid.int_dim(), grid.width(), grid.height(), grid.depth());
	}
	memcpypitch_host_to_device(cgrid.store, cgrid.get_pitch(), grid.store, sizeof(hipDoubleComplex)*grid.width()*grid.height()*grid.depth(), sizeof(hipDoubleComplex)*grid.width()*grid.height()*grid.depth(), grid.int_dim());
	return cgrid;
}

ComplexGrid &copyDeviceToHost (ComplexGrid &grid, const CudaComplexGrid &cgrid)
{
	if ((cgrid.int_dim() != grid.int_dim()) || (cgrid.width() != grid.width()) || (cgrid.height() != grid.height()) || (cgrid.depth() != grid.depth()))
	{
		cout << "Warning resizing ComplexGrid for copying!" << endl;
		grid.resize(cgrid.int_dim(), cgrid.width(), cgrid.height(), cgrid.depth());
	}
	memcpypitch_device_to_host(grid.store, sizeof(hipDoubleComplex)*grid.width()*grid.height()*grid.depth(), cgrid.store, cgrid.get_pitch(), cgrid.width()*cgrid.height()*cgrid.depth()*sizeof(hipDoubleComplex), cgrid.int_dim());
	return grid;
}

#endif
