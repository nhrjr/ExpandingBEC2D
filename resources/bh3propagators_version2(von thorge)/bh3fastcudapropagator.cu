#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
#include <iostream>
//#include <hip/hip_runtime.h>

// includes, kernels
#include "bh3fastcudapropagator.h"
#include <complexgrid.h>
#include <wrapped_cuda_functions.h>

#define BLOCK_1D_LENGTH 512
#define KP_LENGTH 128
#define fix_N 15

// Kernels for r- and k-propagation
#include <hip/hip_complex.h>

using namespace std;

typedef struct {
	hipDoubleComplex kx[KP_LENGTH];
	hipDoubleComplex ky[KP_LENGTH];
	hipDoubleComplex kz[KP_LENGTH];
	double timestepsize;
	double Ut;
	int N;
} KParam;

__constant__ __device__ KParam kp;

__device__ static __inline__ double cuCabs2 (hipDoubleComplex x)
{
	return hipCreal(x)*hipCreal(x) + hipCimag(x)*hipCimag(x);
}

extern __shared__ double base[];

#define result_real(index) base[2 * blockDim.x + index]
#define result_imag(index) base[3 * blockDim.x + index]
#define temp_real(index) base[index]
#define temp_imag(index) base[blockDim.x + index]

__inline__ __device__ void fastprop()
{
	result_real(threadIdx.x) = temp_real(threadIdx.x);
	result_imag(threadIdx.x) = temp_imag(threadIdx.x);
	__syncthreads();
	
	for(int i = 1; i <= kp.N; i++)
	{
		hipDoubleComplex t;
		int ind;
		ind = threadIdx.x - 1;
		if(threadIdx.x == 0)
			ind = blockDim.x - 1;
		t.x = temp_real(ind);
		t.y = temp_imag(ind);
		t.x += -2.0 * temp_real(threadIdx.x);
		t.y += -2.0 * temp_imag(threadIdx.x);
		ind = threadIdx.x + 1;
		if(threadIdx.x == blockDim.x - 1)
			ind = 0;
		t.x += temp_real(ind);
		t.y += temp_imag(ind);
		t.x *= kp.timestepsize / i;
		t.y *= kp.timestepsize / i;
		__syncthreads();
		temp_real(threadIdx.x) = - t.y;
		temp_imag(threadIdx.x) = t.x;
		result_real(threadIdx.x) -= t.y;
		result_imag(threadIdx.x) += t.x;
		__syncthreads();
	}
}

////////////////////////////////////////////////////////////////////////////////
//! propagator in r-space
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void short_fast_zpropagate(hipDoubleComplex *grid)
{
	// grid.z * grid.y * x + grid.z * y + z		: z = threadIdx.x; y = blockIdx.y; x = blockIdx.x;
	// => blockDim.y = 1; blockDim.x = grid.z; gridDim.y = grid.y; gridDim.x = grid.x;
	const unsigned int index = blockDim.x * gridDim.y * blockIdx.x + blockDim.x * blockIdx.y + threadIdx.x;
	
	temp_real(threadIdx.x) = grid[index].x;
	temp_imag(threadIdx.x) = grid[index].y;
	fastprop();
	grid[index].x = result_real(threadIdx.x);
	grid[index].y = result_imag(threadIdx.x);
}

__global__ void short_fast_ypropagate(hipDoubleComplex *grid)
{
	// grid.z * grid.y * x + grid.z * y + z		: z = blockIdx.y; y = threadIdx.x; x = blockIdx.x;
	// => blockDim.y = 1; blockDim.x = grid.y; gridDim.y = grid.z; gridDim.x = grid.x;
	const unsigned int index = gridDim.y * blockDim.x * blockIdx.x + gridDim.y * threadIdx.x + blockIdx.y;
	
	temp_real(threadIdx.x) = grid[index].x;
	temp_imag(threadIdx.x) = grid[index].y;
	fastprop();
	grid[index].x = result_real(threadIdx.x);
	grid[index].y = result_imag(threadIdx.x);
}

__global__ void short_fast_xpropagate(hipDoubleComplex *grid)
{
	// grid.z * grid.y * x + grid.z * y + z		: z = blockIdx.y; y = blockIdx.x; x = threadIdx.x;
	// => blockDim.y = 1; blockDim.x = grid.x; gridDim.y = grid.z; gridDim.x = grid.y;
	const unsigned int index = gridDim.y * gridDim.x * threadIdx.x + gridDim.y * blockIdx.x + blockIdx.y;
	
	temp_real(threadIdx.x) = grid[index].x;
	temp_imag(threadIdx.x) = grid[index].y;
	fastprop();
	// calculate the propagator and propagate
	hipDoubleComplex t;
	hipDoubleComplex t2;
	t.x = result_real(threadIdx.x);
	t.y = result_imag(threadIdx.x);
	t2 = t;
	t.x *= t.x;
	t.y *= t.y;
	t.x += t.y;
	t.x *= kp.Ut;
	sincos(t.x, &t.y, &t.x);
	grid[index] = hipCmul(t, t2);
}

// Bh3CudaPropagator - Class implementation

Bh3FastCudaPropagator::Bh3FastCudaPropagator(const PathOptions &opt, const ComplexGrid &start) :
			Bh3Propagator(opt, start)
{
	if((opt.grid[0] > BLOCK_1D_LENGTH) ||
		(opt.grid[1] > BLOCK_1D_LENGTH) ||
		(opt.grid[2] > BLOCK_1D_LENGTH))
	{
		cout << "Warning: invalid grid-sizes: Must be <= BLOCK_1D_LENGTH=" << BLOCK_1D_LENGTH  << " !" << endl;
	}
	
	dev_rgrid = new CudaComplexGrid(opt.grid[0], opt.grid[1], opt.grid[2]);
	
	KParam p;
	
	p.timestepsize = options.timestepsize;
	p.Ut = - options.U * options.timestepsize;
	p.N = fix_N;
	memcpy_host_to_symbol("kp", &p, sizeof(KParam), 0);
	
	*dev_rgrid = rgrid[0];
}

Bh3FastCudaPropagator::~Bh3FastCudaPropagator()
{
	delete dev_rgrid;
}

bool Bh3FastCudaPropagator::propagate1()
{
	if(options.grid[2] > 1)
	{
		dim3 dimBlock(options.grid[2], 1);
		dim3 dimGrid(options.grid[0],options.grid[1]);
		thread_synchronize("before z");
		short_fast_zpropagate<<<dimGrid, dimBlock, options.grid[2] * 2 * sizeof(hipDoubleComplex)>>>(dev_rgrid->getDevicePointer());
		thread_synchronize("after z");
	}
	if(options.grid[1] > 1)
	{
		dim3 dimBlock(options.grid[1], 1);
		dim3 dimGrid(options.grid[0], options.grid[2]);
		thread_synchronize("before y");
		short_fast_ypropagate<<<dimGrid, dimBlock, options.grid[1] * 2 * sizeof(hipDoubleComplex)>>>(dev_rgrid->getDevicePointer());
		thread_synchronize("after y");
	}
	if(options.grid[0] > 1)
	{
		dim3 dimBlock(options.grid[0], 1);
		dim3 dimGrid(options.grid[1], options.grid[2]);
		thread_synchronize("before x");
		short_fast_xpropagate<<<dimGrid, dimBlock, options.grid[0] * 2 * sizeof(hipDoubleComplex)>>>(dev_rgrid->getDevicePointer());
		thread_synchronize("after x");
	}
	return true;
}

bool Bh3FastCudaPropagator::propagateN(int N)
{
	cout << "Starting propagation to " << N << endl;
	int steps = N;
	for(int n = delta_N.size() - 1; n >= 0; n--)
	{
		steps -= delta_N[n];
		if(steps < 0)
			steps = 0;
		for(int i = 0; i < steps; i++)
		{
			if(!propagate1())
				return false;
		}
		
		rgrid[n] = *dev_rgrid;
		N = steps = N - steps;
	}
	cout << "Finishing propagation to " << N << endl;
	return true;
}
